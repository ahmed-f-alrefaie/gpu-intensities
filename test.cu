#include "hip/hip_runtime.h"
#include "test.cuh"


__host__ void benchmark_half_ls(FintensityJob & intensity,int no_initial_states){


	dipole_initialise(&intensity);
	int nJ=2;
	//The intial state vector
	double* initial_vec = new double[intensity.dimenmax];

	double* gpu_initial_vec=NULL;

	copy_array_to_gpu((void*)initial_vec,(void**)&(gpu_initial_vec),sizeof(double)*intensity.dimenmax,"gpu_initial_vec");
	printf("%p\n",gpu_initial_vec);


	double* corr_vec = new double[intensity.dimenmax*2*intensity.molec.sym_maxdegen];
	double* gpu_corr_vec=NULL;

	copy_array_to_gpu((void*)corr_vec,(void**)&(gpu_corr_vec),sizeof(double)*intensity.dimenmax*2*intensity.molec.sym_maxdegen,"gpu_corr_vec");

	double* half_ls = new double[intensity.dimenmax*2*intensity.molec.sym_maxdegen];
	double* gpu_half_ls;

	copy_array_to_gpu((void*)half_ls,(void**)&(gpu_half_ls),sizeof(double)*intensity.dimenmax*2*intensity.molec.sym_maxdegen,"gpu_half_ls1");


	char filename[1024];
	//Get the filename
	printf("Open vector unit\n");
	FILE** eigenvec_unit = new FILE*[2*intensity.molec.sym_nrepres];
	for(int i =0; i< 2; i++){
		for(int j = 0; j < intensity.molec.sym_nrepres; j++)
		{
			if(intensity.isym_do[j] == false) continue;
			sprintf(filename,j0eigen_vector_gamma_filebase,intensity.jvals[i],j+1);
			printf("Reading %s\n",filename);
			eigenvec_unit[i + j*2] = fopen(filename,"r");
			if(eigenvec_unit[i + j*2] == NULL)
			{
				printf("error opening %s \n",filename);
				exit(0);
			}
		}
	}
	
	//Opened all units, now lets start compuing
	
	//Initialise cublas
	hipblasHandle_t handle;
	hipblasStatus_t stat;
	stat = hipblasCreate(&handle);
	if (stat != HIPBLAS_STATUS_SUCCESS) {
		printf ("CUBLAS initialization failed\n");
		return;
	}
	
	CheckCudaError("Initialisation");
			    // Number of threads in each thread block
    	int blockSize =384;
 
    	// Number of thread blocks in grid
    	int gridSize = (int)ceil((float)intensity.dimenmax/blockSize);


	//Testing variables
	double time=0.0,half_ls_time=0,flipped_half_ls_time=0;
	int states_done = 0;
	hipStream_t* stream = new hipStream_t[2*intensity.molec.sym_maxdegen];
	for(int i =0; i < intensity.molec.sym_maxdegen; i++){
		hipStreamCreate(&stream[i*2]);
		hipStreamCreate(&stream[1 + i*2]);
		}
	printf("Nu_if\tJf Kf quantaF\t <-- \tJI KI tauI quantaI\t Ein_A\tLine_str\n");
	int half_ls_runs =0;
	//Run
	for(int ilevelI = 0; ilevelI < intensity.Neigenlevels; ilevelI++){
	
			    //  ! start measuring time per line

	        int indI = intensity.eigen[ilevelI].jind;
	  //    !
	  //    !dimension of the bases for the initial states
	  //    !
	     int dimenI = intensity.bset_contr[indI+1].Maxcontracts;
	   //   !
	    //  !energy, quanta, and gedeneracy order of the initial state
	    //  !
	      int jI = intensity.eigen[ilevelI].jval;
	      double energyI = intensity.eigen[ilevelI].energy;
	      int igammaI  = intensity.eigen[ilevelI].igamma;
	      int * quantaI = intensity.eigen[ilevelI].quanta;
	      int * normalI = intensity.eigen[ilevelI].normal;
	      int ndegI   = intensity.eigen[ilevelI].ndeg;
	      int nsizeI = intensity.bset_contr[indI+1].nsize[igammaI];

	      FILE* unitI = eigenvec_unit[ indI + (igammaI)*2]; 
	    //   printf("Ilevel = %i\n",ilevelI);

	      if(!energy_filter_lower(intensity,jI,energyI,quantaI)) continue;
	      fseek(unitI,(intensity.eigen[ilevelI].irec[0]-1)*nsizeI*sizeof(double),SEEK_SET);


		//Read vector from file
	    //  printf("Read vector\n");
	     	int tread =  fread(initial_vec,sizeof(double),nsizeI,unitI);

		//for(int i=0; i< nsizeI; i++){
		//	printf("vec[%i]=%16.8e\n",i,initial_vec[i]);}
		//printf("read = %i\n",tread);
		//Transfer it to the GPU
	//	printf("Transfer vector\n");
	        stat = hipblasSetVector(intensity.dimenmax, sizeof(double),initial_vec, 1, gpu_initial_vec, 1);
		CheckCudaError("Set Vector I");

		hipDeviceSynchronize();

	  //    printf("Correlating vectors\n");
		//for(int ideg = 0; ideg < ndegI; ideg++){
		//host_correlate_vectors(&intensity.bset_contr[indI+1],0,igammaI,intensity.bset_contr[indI+1].ijterms,intensity.molec.sym_degen,initial_vec,corr_vec);


 
    	      // Number of thread blocks in grid
              //We have the vector now we compute the half_ls

		device_correlate_vectors<<<gridSize,blockSize>>>(intensity.g_ptrs.bset_contr[indI],0,igammaI, gpu_initial_vec,gpu_corr_vec);

		time = GetTimeMs64();
	       	hipDeviceSynchronize();
		int jF;
			for(int indF =0; indF < nJ; indF++){
				jF=intensity.jvals[indF];
				
				/*					do ilevelF = 1, nlevels
				  !
				  if (eigen(ilevelF)%jval/=jF) cycle 
				  !
				  energyF = eigen(ilevelF)%energy
				  igammaF = eigen(ilevelF)%igamma        
				  quantaF(0:nmodes) = eigen(ilevelF)%quanta(0:nmodes) 
				  !
				  call intens_filter(jI,jF,energyI,energyF,igammaI,igammaF,quantaI,quantaF,igamma_pair,passed)
				  !
				  if (passed) exit
				  !
				  !passed = passed_
				  !
				enddo
				*/
				if(!indF_filter(intensity,jI,jF,energyI,igammaI,quantaI))continue;
     	
			for(int ideg=0; ideg < ndegI; ideg++){
					half_ls_runs++;
				do_1st_half_ls(intensity.g_ptrs.bset_contr[indI],intensity.g_ptrs.bset_contr[indF],intensity.dimenmax,ideg,igammaI,intensity.g_ptrs.dipole_me
							, gpu_initial_vec,gpu_corr_vec+ indF*intensity.dimenmax + ideg*intensity.dimenmax*nJ,intensity.g_ptrs.threej,gpu_half_ls + indF*intensity.dimenmax + ideg*intensity.dimenmax*nJ,stream[indF + ideg*nJ]);				
			}
	      	}
		hipDeviceSynchronize();
		CheckCudaError("Flipped half ls");

		time = GetTimeMs64()-time;
		flipped_half_ls_time += time/1000.0;	
		printf("%i - Flipped half_ls done in: %11.4fs\n",states_done,time/1000.0);
/*		
		time = GetTimeMs64();

	       hipDeviceSynchronize();
	       for(int ideg=0; ideg < ndegI; ideg++){
			for(int indF =0; indF < nJ; indF++){
		
				do_1st_half_ls_branch(intensity.g_ptrs.bset_contr[indI],intensity.g_ptrs.bset_contr[indF],intensity.dimenmax,ideg,igammaI,intensity.g_ptrs.dipole_me
							, gpu_initial_vec,gpu_corr_vec+ indF*intensity.dimenmax + ideg*intensity.dimenmax*nJ,intensity.g_ptrs.threej,gpu_half_ls + indF*intensity.dimenmax + ideg*intensity.dimenmax*nJ,stream[indF + ideg*nJ]);				
			}
	      }
		hipDeviceSynchronize();
		CheckCudaError("Branch");

		time = GetTimeMs64()-time;
		printf("%i - Branch half_ls done in: %11.4fs\n",states_done,time/1000.0);
		half_ls_time += time/1000.0;
*/		
		

		CheckCudaError("First run");
		states_done++;
		if(states_done >= no_initial_states) break;

		
		
		


	}
		printf("State stats-  largest dimension: %i number of degeneracies: %i\n",intensity.dimenmax,intensity.molec.sym_maxdegen) ;
		printf("-----------------Time results---------------------\n");
		printf("Branch - Total time: %11.4fs Average Time per state: %11.4fs\n",half_ls_time,half_ls_time/double(no_initial_states));
		printf("Flipped - Total time: %11.4fs Average Time per state: %11.4fs Average time per call = %11.4f\n",flipped_half_ls_time,flipped_half_ls_time/double(no_initial_states) ,flipped_half_ls_time/(double(no_initial_states)*half_ls_runs));
		printf("Total states completed: %i\n",states_done);
	
		hipDeviceReset();

	

		
	
};

